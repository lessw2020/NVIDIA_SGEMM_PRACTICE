// basic matmul


#include <hip/hip_runtime.h>
#include <random>

__global__ void matmul (float* a, float* b, float* c, int N) {

}

void init_matrix(float* m, int N) {
    // Create a random number generator with a fixed seed
    std::mt19937 rng(2020); // Mersenne Twister

    // Create a distribution for generating random floats between 0 and 100
    std::uniform_real_distribution<float> dist(0.0f, 100.0f);
    int total_size = N*N;

    for (int i=0; i< total_size; ++i) {
        m[i] = dist(rng);

    }
}

int main() {
// square matrix assumed (2 ^ 10 * 2 ^ 10 default)
int N = 1 << 10;

size_t bytes = N * N * sizeof(float);
float *a, *b, *c;
hipMallocManaged(&a, bytes);
hipMallocManaged(&b, bytes);
hipMallocManaged(&c, bytes);

//init matrices
init_matrix(a,N);
init_matrix(b,N);

// set block and grid dims
int threads = 16;
int blocks = (N + threads -1) / threads;

dim3 Threads (threads, threads);
dim3 Blocks (blocks, blocks);

matmul<<<Blocks, Threads>>>(a, b, c, N);
hipDeviceSynchronize();






}
