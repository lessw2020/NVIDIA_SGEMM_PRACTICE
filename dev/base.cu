#include "hip/hip_runtime.h"
// basic matmul

#include <random>

__global__ void matmul () {

}

void init_matrix(float* m, int N) {
    // Create a random number generator with a fixed seed
    std::mt19937 rng(2020); // Mersenne Twister

    // Create a distribution for generating random floats between 0 and 100
    std::uniform_real_distribution<float> dist(0.0f, 100.0f);

    for (i=0; i<N; ++i) {
        m[i] = dist(rng)

    }
}

int main() {
// square matrix assumed (2 ^ 10 * 2 ^ 10 default)
int N = 1 << 10;

size_t bytes = N * N * sizeof(float);
float *a, *b, *c;
hipMallocManaged(&a, bytes);
hipMallocManaged(&b, bytes);
hipMallocManaged(&c, bytes);

//init matrices
init_matrix(a,N);
init_matrix(b,N);







}
